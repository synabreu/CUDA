#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>


#define N_THREADS 512

__global__ void addByThreads(int *a, int *b, int *c)
{
	// a block can be split into parallel threads.
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i = 0; i<size; i++) {
		x[i] = rand() % 10;
	}
}


int main()
{
	int *host_a, *host_b, *host_c;    // host copies of host_a, host_b, host_c
	int *device_a, *device_b, *device_c; // device copies of device_a, device_b, device_c

	int size = N_THREADS * sizeof(int);

	// Alloc space for device copies of device_a, device_b, device_c
	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMalloc(&device_c, size);

	// Alloc space for host copies of host_a, host_b, host_c 
	// and setup input values
	host_a = (int*)malloc(size);
	random_ints(host_a, N_THREADS);

	host_b = (int*)malloc(size);
	random_ints(host_b, N_THREADS);

	host_c = (int*)malloc(size);

	// Copy input to device
	hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
	hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N threads
	addByThreads << <1, N_THREADS >> > (device_a, device_b, device_c);

	// Copy result back to host
	hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i<N_THREADS; i++) {
		printf("host_a[%d]=%d , host_b[%d]=%d, host_c[%d]=%d\n", i, host_a[i], i, host_b[i], i, host_c[i]);
	}

	// Cleanup
	free(host_a); free(host_b); free(host_c);
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

    return 0;
}


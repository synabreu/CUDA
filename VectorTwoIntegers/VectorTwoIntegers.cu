
#include "hip/hip_runtime.h"


#include <stdio.h>

// In device's code, use pointers for the variables, 
// which means parameters in the addTwoIntegersKernel function. 
// So a, b, and c must point to device memory and then allocate memory on GPU internally.
__global__ void addTwoIntegers(int *a, int *b, int *c)
{
//	int i = threadIdx.x;
	*c = *a + *b;
}

int main(void)
{
	int host_a, host_b, host_c; // host copies of a, b, c
	int *device_a, *device_b, *device_c; // device copies of a, b, c
	int size = sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&device_a, size);
	hipMalloc((void **)&device_b, size);
	hipMalloc((void **)&device_c, size);

	// Setup input values
	host_a = 10;
	host_b = 19;

	// copy inputs to device
	hipMemcpy(device_a, &host_a, size, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &host_b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	addTwoIntegers<<<1,1>>>(device_a, device_b, device_c);
	
	// Copy result back to host
	hipMemcpy(&host_c, device_c, size, hipMemcpyDeviceToHost);
	
	// debug code
	printf("%d\n", host_a + host_b);
	printf("%d\n", host_c);
	printf("%d\n", device_c);

	// Cleanup
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
    return 0;
}

